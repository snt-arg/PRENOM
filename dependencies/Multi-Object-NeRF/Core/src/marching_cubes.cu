#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

/** @file   marching_cubes.h
 *  @author Thomas Müller & Alex Evans, NVIDIA
 */

/** Modification
* @date 12/06/2022
* @author Xiao Han
*/

#include "marching_cubes.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include <third_party/tiny-cuda-nn/dependencies/stbi/stb_image_write.h>

namespace nerf{

Eigen::Vector3i GetMarchingCubesRes(uint32_t res_1d, const BoundingBox& aabb)
{   
    /* float scale = res_1d / (aabb.max - aabb.min).maxCoeff();
	Eigen::Vector3i res3i = ((aabb.max - aabb.min) * scale + Eigen::Vector3f::Constant(0.5f)).cast<int>();
	res3i.x() = tcnn::next_multiple((unsigned int)res3i.x(), 16u);
	res3i.y() = tcnn::next_multiple((unsigned int)res3i.y(), 16u);
	res3i.z() = tcnn::next_multiple((unsigned int)res3i.z(), 16u); */
	
	Eigen::Vector3i res3i(res_1d,res_1d,res_1d);
	
	return res3i;

}

__global__ void gen_vertices(BoundingBox aabb, Eigen::Vector3i res3i, const float* __restrict__ density, int*__restrict__ vertidx_grid, Eigen::Vector3f* verts_out, float thresh, uint32_t* __restrict__ counters) 
{
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t z = blockIdx.z * blockDim.z + threadIdx.z;
	if (x>=res3i.x() || y>=res3i.y() || z>=res3i.z()) 
		return;
	Eigen::Vector3f scale = (aabb.max-aabb.min).cwiseQuotient((res3i-Eigen::Vector3i::Ones()).cast<float>());
	Eigen::Vector3f offset = aabb.min;
	uint32_t res2 = res3i.x() * res3i.y();
	uint32_t res3 = res3i.x() * res3i.y() * res3i.z();
	uint32_t idx = x + y * res3i.x() + z * res2;
	float f0 = density[idx];
	bool inside=(f0>thresh);
	if (x<res3i.x()-1) {
		float f1 = density[idx+1];
		if (inside != (f1>thresh)) {
			uint32_t vidx = atomicAdd(counters,1);
			if (verts_out) {
				vertidx_grid[idx]=vidx+1;
				float prevf=f0,nextf=f1;
				float dt=((thresh-prevf)/(nextf-prevf));
				verts_out[vidx] = Eigen::Vector3f{float(x)+dt, float(y), float(z)}.cwiseProduct(scale) + offset;
			}
		}
	}
	if (y<res3i.y()-1) {
		float f1 = density[idx+res3i.x()];
		if (inside != (f1>thresh)) {
			uint32_t vidx = atomicAdd(counters,1);
			if (verts_out) {
				vertidx_grid[idx+res3]=vidx+1;
				float prevf=f0,nextf=f1;
				float dt=((thresh-prevf)/(nextf-prevf));
				verts_out[vidx]= Eigen::Vector3f{float(x), float(y)+dt, float(z)}.cwiseProduct(scale) + offset;
			}
		}
	}
	if (z<res3i.z()-1) {
		float f1 = density[idx+res2];
		if (inside != (f1>thresh)) {
			uint32_t vidx = atomicAdd(counters,1);
			if (verts_out) {
				vertidx_grid[idx+res3*2]=vidx+1;
				float prevf=f0,nextf=f1;
				float dt=((thresh-prevf)/(nextf-prevf));
				verts_out[vidx]= Eigen::Vector3f{float(x), float(y), float(z)+dt}.cwiseProduct(scale) + offset;
			}
		}
	}
}

__global__ void gen_faces(Eigen::Vector3i res3i, const float* __restrict__ density, const int*__restrict__ vertidx_grid, uint32_t* indices_out, float thresh, uint32_t *__restrict__ counters) 
{
	// marching cubes tables from https://github.com/pmneila/PyMCubes/blob/master/mcubes/src/marchingcubes.cpp which in turn seems to be from https://web.archive.org/web/20181127124338/http://paulbourke.net/geometry/polygonise/
	// License is BSD 3-clause, which can be found here: https://github.com/pmneila/PyMCubes/blob/master/LICENSE
	/*
	static constexpr uint16_t edge_table[256] =
	{
		0x000, 0x109, 0x203, 0x30a, 0x406, 0x50f, 0x605, 0x70c, 0x80c, 0x905, 0xa0f, 0xb06, 0xc0a, 0xd03, 0xe09, 0xf00,
		0x190, 0x099, 0x393, 0x29a, 0x596, 0x49f, 0x795, 0x69c, 0x99c, 0x895, 0xb9f, 0xa96, 0xd9a, 0xc93, 0xf99, 0xe90,
		0x230, 0x339, 0x033, 0x13a, 0x636, 0x73f, 0x435, 0x53c, 0xa3c, 0xb35, 0x83f, 0x936, 0xe3a, 0xf33, 0xc39, 0xd30,
		0x3a0, 0x2a9, 0x1a3, 0x0aa, 0x7a6, 0x6af, 0x5a5, 0x4ac, 0xbac, 0xaa5, 0x9af, 0x8a6, 0xfaa, 0xea3, 0xda9, 0xca0,
		0x460, 0x569, 0x663, 0x76a, 0x066, 0x16f, 0x265, 0x36c, 0xc6c, 0xd65, 0xe6f, 0xf66, 0x86a, 0x963, 0xa69, 0xb60,
		0x5f0, 0x4f9, 0x7f3, 0x6fa, 0x1f6, 0x0ff, 0x3f5, 0x2fc, 0xdfc, 0xcf5, 0xfff, 0xef6, 0x9fa, 0x8f3, 0xbf9, 0xaf0,
		0x650, 0x759, 0x453, 0x55a, 0x256, 0x35f, 0x055, 0x15c, 0xe5c, 0xf55, 0xc5f, 0xd56, 0xa5a, 0xb53, 0x859, 0x950,
		0x7c0, 0x6c9, 0x5c3, 0x4ca, 0x3c6, 0x2cf, 0x1c5, 0x0cc, 0xfcc, 0xec5, 0xdcf, 0xcc6, 0xbca, 0xac3, 0x9c9, 0x8c0,
		0x8c0, 0x9c9, 0xac3, 0xbca, 0xcc6, 0xdcf, 0xec5, 0xfcc, 0x0cc, 0x1c5, 0x2cf, 0x3c6, 0x4ca, 0x5c3, 0x6c9, 0x7c0,
		0x950, 0x859, 0xb53, 0xa5a, 0xd56, 0xc5f, 0xf55, 0xe5c, 0x15c, 0x055, 0x35f, 0x256, 0x55a, 0x453, 0x759, 0x650,
		0xaf0, 0xbf9, 0x8f3, 0x9fa, 0xef6, 0xfff, 0xcf5, 0xdfc, 0x2fc, 0x3f5, 0x0ff, 0x1f6, 0x6fa, 0x7f3, 0x4f9, 0x5f0,
		0xb60, 0xa69, 0x963, 0x86a, 0xf66, 0xe6f, 0xd65, 0xc6c, 0x36c, 0x265, 0x16f, 0x066, 0x76a, 0x663, 0x569, 0x460,
		0xca0, 0xda9, 0xea3, 0xfaa, 0x8a6, 0x9af, 0xaa5, 0xbac, 0x4ac, 0x5a5, 0x6af, 0x7a6, 0x0aa, 0x1a3, 0x2a9, 0x3a0,
		0xd30, 0xc39, 0xf33, 0xe3a, 0x936, 0x83f, 0xb35, 0xa3c, 0x53c, 0x435, 0x73f, 0x636, 0x13a, 0x033, 0x339, 0x230,
		0xe90, 0xf99, 0xc93, 0xd9a, 0xa96, 0xb9f, 0x895, 0x99c, 0x69c, 0x795, 0x49f, 0x596, 0x29a, 0x393, 0x099, 0x190,
		0xf00, 0xe09, 0xd03, 0xc0a, 0xb06, 0xa0f, 0x905, 0x80c, 0x70c, 0x605, 0x50f, 0x406, 0x30a, 0x203, 0x109, 0x000
	};
	*/
	static constexpr int8_t triangle_table[256][16] =
	{
		{-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{0, 8, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{0, 1, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{1, 8, 3, 9, 8, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{1, 2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{0, 8, 3, 1, 2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{9, 2, 10, 0, 2, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{2, 8, 3, 2, 10, 8, 10, 9, 8, -1, -1, -1, -1, -1, -1, -1},
		{3, 11, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{0, 11, 2, 8, 11, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{1, 9, 0, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{1, 11, 2, 1, 9, 11, 9, 8, 11, -1, -1, -1, -1, -1, -1, -1},
		{3, 10, 1, 11, 10, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{0, 10, 1, 0, 8, 10, 8, 11, 10, -1, -1, -1, -1, -1, -1, -1},
		{3, 9, 0, 3, 11, 9, 11, 10, 9, -1, -1, -1, -1, -1, -1, -1},
		{9, 8, 10, 10, 8, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{4, 7, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{4, 3, 0, 7, 3, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{0, 1, 9, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{4, 1, 9, 4, 7, 1, 7, 3, 1, -1, -1, -1, -1, -1, -1, -1},
		{1, 2, 10, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{3, 4, 7, 3, 0, 4, 1, 2, 10, -1, -1, -1, -1, -1, -1, -1},
		{9, 2, 10, 9, 0, 2, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1},
		{2, 10, 9, 2, 9, 7, 2, 7, 3, 7, 9, 4, -1, -1, -1, -1},
		{8, 4, 7, 3, 11, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{11, 4, 7, 11, 2, 4, 2, 0, 4, -1, -1, -1, -1, -1, -1, -1},
		{9, 0, 1, 8, 4, 7, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1},
		{4, 7, 11, 9, 4, 11, 9, 11, 2, 9, 2, 1, -1, -1, -1, -1},
		{3, 10, 1, 3, 11, 10, 7, 8, 4, -1, -1, -1, -1, -1, -1, -1},
		{1, 11, 10, 1, 4, 11, 1, 0, 4, 7, 11, 4, -1, -1, -1, -1},
		{4, 7, 8, 9, 0, 11, 9, 11, 10, 11, 0, 3, -1, -1, -1, -1},
		{4, 7, 11, 4, 11, 9, 9, 11, 10, -1, -1, -1, -1, -1, -1, -1},
		{9, 5, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{9, 5, 4, 0, 8, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{0, 5, 4, 1, 5, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{8, 5, 4, 8, 3, 5, 3, 1, 5, -1, -1, -1, -1, -1, -1, -1},
		{1, 2, 10, 9, 5, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{3, 0, 8, 1, 2, 10, 4, 9, 5, -1, -1, -1, -1, -1, -1, -1},
		{5, 2, 10, 5, 4, 2, 4, 0, 2, -1, -1, -1, -1, -1, -1, -1},
		{2, 10, 5, 3, 2, 5, 3, 5, 4, 3, 4, 8, -1, -1, -1, -1},
		{9, 5, 4, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{0, 11, 2, 0, 8, 11, 4, 9, 5, -1, -1, -1, -1, -1, -1, -1},
		{0, 5, 4, 0, 1, 5, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1},
		{2, 1, 5, 2, 5, 8, 2, 8, 11, 4, 8, 5, -1, -1, -1, -1},
		{10, 3, 11, 10, 1, 3, 9, 5, 4, -1, -1, -1, -1, -1, -1, -1},
		{4, 9, 5, 0, 8, 1, 8, 10, 1, 8, 11, 10, -1, -1, -1, -1},
		{5, 4, 0, 5, 0, 11, 5, 11, 10, 11, 0, 3, -1, -1, -1, -1},
		{5, 4, 8, 5, 8, 10, 10, 8, 11, -1, -1, -1, -1, -1, -1, -1},
		{9, 7, 8, 5, 7, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{9, 3, 0, 9, 5, 3, 5, 7, 3, -1, -1, -1, -1, -1, -1, -1},
		{0, 7, 8, 0, 1, 7, 1, 5, 7, -1, -1, -1, -1, -1, -1, -1},
		{1, 5, 3, 3, 5, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{9, 7, 8, 9, 5, 7, 10, 1, 2, -1, -1, -1, -1, -1, -1, -1},
		{10, 1, 2, 9, 5, 0, 5, 3, 0, 5, 7, 3, -1, -1, -1, -1},
		{8, 0, 2, 8, 2, 5, 8, 5, 7, 10, 5, 2, -1, -1, -1, -1},
		{2, 10, 5, 2, 5, 3, 3, 5, 7, -1, -1, -1, -1, -1, -1, -1},
		{7, 9, 5, 7, 8, 9, 3, 11, 2, -1, -1, -1, -1, -1, -1, -1},
		{9, 5, 7, 9, 7, 2, 9, 2, 0, 2, 7, 11, -1, -1, -1, -1},
		{2, 3, 11, 0, 1, 8, 1, 7, 8, 1, 5, 7, -1, -1, -1, -1},
		{11, 2, 1, 11, 1, 7, 7, 1, 5, -1, -1, -1, -1, -1, -1, -1},
		{9, 5, 8, 8, 5, 7, 10, 1, 3, 10, 3, 11, -1, -1, -1, -1},
		{5, 7, 0, 5, 0, 9, 7, 11, 0, 1, 0, 10, 11, 10, 0, -1},
		{11, 10, 0, 11, 0, 3, 10, 5, 0, 8, 0, 7, 5, 7, 0, -1},
		{11, 10, 5, 7, 11, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{10, 6, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{0, 8, 3, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{9, 0, 1, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{1, 8, 3, 1, 9, 8, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1},
		{1, 6, 5, 2, 6, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{1, 6, 5, 1, 2, 6, 3, 0, 8, -1, -1, -1, -1, -1, -1, -1},
		{9, 6, 5, 9, 0, 6, 0, 2, 6, -1, -1, -1, -1, -1, -1, -1},
		{5, 9, 8, 5, 8, 2, 5, 2, 6, 3, 2, 8, -1, -1, -1, -1},
		{2, 3, 11, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{11, 0, 8, 11, 2, 0, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1},
		{0, 1, 9, 2, 3, 11, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1},
		{5, 10, 6, 1, 9, 2, 9, 11, 2, 9, 8, 11, -1, -1, -1, -1},
		{6, 3, 11, 6, 5, 3, 5, 1, 3, -1, -1, -1, -1, -1, -1, -1},
		{0, 8, 11, 0, 11, 5, 0, 5, 1, 5, 11, 6, -1, -1, -1, -1},
		{3, 11, 6, 0, 3, 6, 0, 6, 5, 0, 5, 9, -1, -1, -1, -1},
		{6, 5, 9, 6, 9, 11, 11, 9, 8, -1, -1, -1, -1, -1, -1, -1},
		{5, 10, 6, 4, 7, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{4, 3, 0, 4, 7, 3, 6, 5, 10, -1, -1, -1, -1, -1, -1, -1},
		{1, 9, 0, 5, 10, 6, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1},
		{10, 6, 5, 1, 9, 7, 1, 7, 3, 7, 9, 4, -1, -1, -1, -1},
		{6, 1, 2, 6, 5, 1, 4, 7, 8, -1, -1, -1, -1, -1, -1, -1},
		{1, 2, 5, 5, 2, 6, 3, 0, 4, 3, 4, 7, -1, -1, -1, -1},
		{8, 4, 7, 9, 0, 5, 0, 6, 5, 0, 2, 6, -1, -1, -1, -1},
		{7, 3, 9, 7, 9, 4, 3, 2, 9, 5, 9, 6, 2, 6, 9, -1},
		{3, 11, 2, 7, 8, 4, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1},
		{5, 10, 6, 4, 7, 2, 4, 2, 0, 2, 7, 11, -1, -1, -1, -1},
		{0, 1, 9, 4, 7, 8, 2, 3, 11, 5, 10, 6, -1, -1, -1, -1},
		{9, 2, 1, 9, 11, 2, 9, 4, 11, 7, 11, 4, 5, 10, 6, -1},
		{8, 4, 7, 3, 11, 5, 3, 5, 1, 5, 11, 6, -1, -1, -1, -1},
		{5, 1, 11, 5, 11, 6, 1, 0, 11, 7, 11, 4, 0, 4, 11, -1},
		{0, 5, 9, 0, 6, 5, 0, 3, 6, 11, 6, 3, 8, 4, 7, -1},
		{6, 5, 9, 6, 9, 11, 4, 7, 9, 7, 11, 9, -1, -1, -1, -1},
		{10, 4, 9, 6, 4, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{4, 10, 6, 4, 9, 10, 0, 8, 3, -1, -1, -1, -1, -1, -1, -1},
		{10, 0, 1, 10, 6, 0, 6, 4, 0, -1, -1, -1, -1, -1, -1, -1},
		{8, 3, 1, 8, 1, 6, 8, 6, 4, 6, 1, 10, -1, -1, -1, -1},
		{1, 4, 9, 1, 2, 4, 2, 6, 4, -1, -1, -1, -1, -1, -1, -1},
		{3, 0, 8, 1, 2, 9, 2, 4, 9, 2, 6, 4, -1, -1, -1, -1},
		{0, 2, 4, 4, 2, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{8, 3, 2, 8, 2, 4, 4, 2, 6, -1, -1, -1, -1, -1, -1, -1},
		{10, 4, 9, 10, 6, 4, 11, 2, 3, -1, -1, -1, -1, -1, -1, -1},
		{0, 8, 2, 2, 8, 11, 4, 9, 10, 4, 10, 6, -1, -1, -1, -1},
		{3, 11, 2, 0, 1, 6, 0, 6, 4, 6, 1, 10, -1, -1, -1, -1},
		{6, 4, 1, 6, 1, 10, 4, 8, 1, 2, 1, 11, 8, 11, 1, -1},
		{9, 6, 4, 9, 3, 6, 9, 1, 3, 11, 6, 3, -1, -1, -1, -1},
		{8, 11, 1, 8, 1, 0, 11, 6, 1, 9, 1, 4, 6, 4, 1, -1},
		{3, 11, 6, 3, 6, 0, 0, 6, 4, -1, -1, -1, -1, -1, -1, -1},
		{6, 4, 8, 11, 6, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{7, 10, 6, 7, 8, 10, 8, 9, 10, -1, -1, -1, -1, -1, -1, -1},
		{0, 7, 3, 0, 10, 7, 0, 9, 10, 6, 7, 10, -1, -1, -1, -1},
		{10, 6, 7, 1, 10, 7, 1, 7, 8, 1, 8, 0, -1, -1, -1, -1},
		{10, 6, 7, 10, 7, 1, 1, 7, 3, -1, -1, -1, -1, -1, -1, -1},
		{1, 2, 6, 1, 6, 8, 1, 8, 9, 8, 6, 7, -1, -1, -1, -1},
		{2, 6, 9, 2, 9, 1, 6, 7, 9, 0, 9, 3, 7, 3, 9, -1},
		{7, 8, 0, 7, 0, 6, 6, 0, 2, -1, -1, -1, -1, -1, -1, -1},
		{7, 3, 2, 6, 7, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{2, 3, 11, 10, 6, 8, 10, 8, 9, 8, 6, 7, -1, -1, -1, -1},
		{2, 0, 7, 2, 7, 11, 0, 9, 7, 6, 7, 10, 9, 10, 7, -1},
		{1, 8, 0, 1, 7, 8, 1, 10, 7, 6, 7, 10, 2, 3, 11, -1},
		{11, 2, 1, 11, 1, 7, 10, 6, 1, 6, 7, 1, -1, -1, -1, -1},
		{8, 9, 6, 8, 6, 7, 9, 1, 6, 11, 6, 3, 1, 3, 6, -1},
		{0, 9, 1, 11, 6, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{7, 8, 0, 7, 0, 6, 3, 11, 0, 11, 6, 0, -1, -1, -1, -1},
		{7, 11, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{7, 6, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{3, 0, 8, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{0, 1, 9, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{8, 1, 9, 8, 3, 1, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1},
		{10, 1, 2, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{1, 2, 10, 3, 0, 8, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1},
		{2, 9, 0, 2, 10, 9, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1},
		{6, 11, 7, 2, 10, 3, 10, 8, 3, 10, 9, 8, -1, -1, -1, -1},
		{7, 2, 3, 6, 2, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{7, 0, 8, 7, 6, 0, 6, 2, 0, -1, -1, -1, -1, -1, -1, -1},
		{2, 7, 6, 2, 3, 7, 0, 1, 9, -1, -1, -1, -1, -1, -1, -1},
		{1, 6, 2, 1, 8, 6, 1, 9, 8, 8, 7, 6, -1, -1, -1, -1},
		{10, 7, 6, 10, 1, 7, 1, 3, 7, -1, -1, -1, -1, -1, -1, -1},
		{10, 7, 6, 1, 7, 10, 1, 8, 7, 1, 0, 8, -1, -1, -1, -1},
		{0, 3, 7, 0, 7, 10, 0, 10, 9, 6, 10, 7, -1, -1, -1, -1},
		{7, 6, 10, 7, 10, 8, 8, 10, 9, -1, -1, -1, -1, -1, -1, -1},
		{6, 8, 4, 11, 8, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{3, 6, 11, 3, 0, 6, 0, 4, 6, -1, -1, -1, -1, -1, -1, -1},
		{8, 6, 11, 8, 4, 6, 9, 0, 1, -1, -1, -1, -1, -1, -1, -1},
		{9, 4, 6, 9, 6, 3, 9, 3, 1, 11, 3, 6, -1, -1, -1, -1},
		{6, 8, 4, 6, 11, 8, 2, 10, 1, -1, -1, -1, -1, -1, -1, -1},
		{1, 2, 10, 3, 0, 11, 0, 6, 11, 0, 4, 6, -1, -1, -1, -1},
		{4, 11, 8, 4, 6, 11, 0, 2, 9, 2, 10, 9, -1, -1, -1, -1},
		{10, 9, 3, 10, 3, 2, 9, 4, 3, 11, 3, 6, 4, 6, 3, -1},
		{8, 2, 3, 8, 4, 2, 4, 6, 2, -1, -1, -1, -1, -1, -1, -1},
		{0, 4, 2, 4, 6, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{1, 9, 0, 2, 3, 4, 2, 4, 6, 4, 3, 8, -1, -1, -1, -1},
		{1, 9, 4, 1, 4, 2, 2, 4, 6, -1, -1, -1, -1, -1, -1, -1},
		{8, 1, 3, 8, 6, 1, 8, 4, 6, 6, 10, 1, -1, -1, -1, -1},
		{10, 1, 0, 10, 0, 6, 6, 0, 4, -1, -1, -1, -1, -1, -1, -1},
		{4, 6, 3, 4, 3, 8, 6, 10, 3, 0, 3, 9, 10, 9, 3, -1},
		{10, 9, 4, 6, 10, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{4, 9, 5, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{0, 8, 3, 4, 9, 5, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1},
		{5, 0, 1, 5, 4, 0, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1},
		{11, 7, 6, 8, 3, 4, 3, 5, 4, 3, 1, 5, -1, -1, -1, -1},
		{9, 5, 4, 10, 1, 2, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1},
		{6, 11, 7, 1, 2, 10, 0, 8, 3, 4, 9, 5, -1, -1, -1, -1},
		{7, 6, 11, 5, 4, 10, 4, 2, 10, 4, 0, 2, -1, -1, -1, -1},
		{3, 4, 8, 3, 5, 4, 3, 2, 5, 10, 5, 2, 11, 7, 6, -1},
		{7, 2, 3, 7, 6, 2, 5, 4, 9, -1, -1, -1, -1, -1, -1, -1},
		{9, 5, 4, 0, 8, 6, 0, 6, 2, 6, 8, 7, -1, -1, -1, -1},
		{3, 6, 2, 3, 7, 6, 1, 5, 0, 5, 4, 0, -1, -1, -1, -1},
		{6, 2, 8, 6, 8, 7, 2, 1, 8, 4, 8, 5, 1, 5, 8, -1},
		{9, 5, 4, 10, 1, 6, 1, 7, 6, 1, 3, 7, -1, -1, -1, -1},
		{1, 6, 10, 1, 7, 6, 1, 0, 7, 8, 7, 0, 9, 5, 4, -1},
		{4, 0, 10, 4, 10, 5, 0, 3, 10, 6, 10, 7, 3, 7, 10, -1},
		{7, 6, 10, 7, 10, 8, 5, 4, 10, 4, 8, 10, -1, -1, -1, -1},
		{6, 9, 5, 6, 11, 9, 11, 8, 9, -1, -1, -1, -1, -1, -1, -1},
		{3, 6, 11, 0, 6, 3, 0, 5, 6, 0, 9, 5, -1, -1, -1, -1},
		{0, 11, 8, 0, 5, 11, 0, 1, 5, 5, 6, 11, -1, -1, -1, -1},
		{6, 11, 3, 6, 3, 5, 5, 3, 1, -1, -1, -1, -1, -1, -1, -1},
		{1, 2, 10, 9, 5, 11, 9, 11, 8, 11, 5, 6, -1, -1, -1, -1},
		{0, 11, 3, 0, 6, 11, 0, 9, 6, 5, 6, 9, 1, 2, 10, -1},
		{11, 8, 5, 11, 5, 6, 8, 0, 5, 10, 5, 2, 0, 2, 5, -1},
		{6, 11, 3, 6, 3, 5, 2, 10, 3, 10, 5, 3, -1, -1, -1, -1},
		{5, 8, 9, 5, 2, 8, 5, 6, 2, 3, 8, 2, -1, -1, -1, -1},
		{9, 5, 6, 9, 6, 0, 0, 6, 2, -1, -1, -1, -1, -1, -1, -1},
		{1, 5, 8, 1, 8, 0, 5, 6, 8, 3, 8, 2, 6, 2, 8, -1},
		{1, 5, 6, 2, 1, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{1, 3, 6, 1, 6, 10, 3, 8, 6, 5, 6, 9, 8, 9, 6, -1},
		{10, 1, 0, 10, 0, 6, 9, 5, 0, 5, 6, 0, -1, -1, -1, -1},
		{0, 3, 8, 5, 6, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{10, 5, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{11, 5, 10, 7, 5, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{11, 5, 10, 11, 7, 5, 8, 3, 0, -1, -1, -1, -1, -1, -1, -1},
		{5, 11, 7, 5, 10, 11, 1, 9, 0, -1, -1, -1, -1, -1, -1, -1},
		{10, 7, 5, 10, 11, 7, 9, 8, 1, 8, 3, 1, -1, -1, -1, -1},
		{11, 1, 2, 11, 7, 1, 7, 5, 1, -1, -1, -1, -1, -1, -1, -1},
		{0, 8, 3, 1, 2, 7, 1, 7, 5, 7, 2, 11, -1, -1, -1, -1},
		{9, 7, 5, 9, 2, 7, 9, 0, 2, 2, 11, 7, -1, -1, -1, -1},
		{7, 5, 2, 7, 2, 11, 5, 9, 2, 3, 2, 8, 9, 8, 2, -1},
		{2, 5, 10, 2, 3, 5, 3, 7, 5, -1, -1, -1, -1, -1, -1, -1},
		{8, 2, 0, 8, 5, 2, 8, 7, 5, 10, 2, 5, -1, -1, -1, -1},
		{9, 0, 1, 5, 10, 3, 5, 3, 7, 3, 10, 2, -1, -1, -1, -1},
		{9, 8, 2, 9, 2, 1, 8, 7, 2, 10, 2, 5, 7, 5, 2, -1},
		{1, 3, 5, 3, 7, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{0, 8, 7, 0, 7, 1, 1, 7, 5, -1, -1, -1, -1, -1, -1, -1},
		{9, 0, 3, 9, 3, 5, 5, 3, 7, -1, -1, -1, -1, -1, -1, -1},
		{9, 8, 7, 5, 9, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{5, 8, 4, 5, 10, 8, 10, 11, 8, -1, -1, -1, -1, -1, -1, -1},
		{5, 0, 4, 5, 11, 0, 5, 10, 11, 11, 3, 0, -1, -1, -1, -1},
		{0, 1, 9, 8, 4, 10, 8, 10, 11, 10, 4, 5, -1, -1, -1, -1},
		{10, 11, 4, 10, 4, 5, 11, 3, 4, 9, 4, 1, 3, 1, 4, -1},
		{2, 5, 1, 2, 8, 5, 2, 11, 8, 4, 5, 8, -1, -1, -1, -1},
		{0, 4, 11, 0, 11, 3, 4, 5, 11, 2, 11, 1, 5, 1, 11, -1},
		{0, 2, 5, 0, 5, 9, 2, 11, 5, 4, 5, 8, 11, 8, 5, -1},
		{9, 4, 5, 2, 11, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{2, 5, 10, 3, 5, 2, 3, 4, 5, 3, 8, 4, -1, -1, -1, -1},
		{5, 10, 2, 5, 2, 4, 4, 2, 0, -1, -1, -1, -1, -1, -1, -1},
		{3, 10, 2, 3, 5, 10, 3, 8, 5, 4, 5, 8, 0, 1, 9, -1},
		{5, 10, 2, 5, 2, 4, 1, 9, 2, 9, 4, 2, -1, -1, -1, -1},
		{8, 4, 5, 8, 5, 3, 3, 5, 1, -1, -1, -1, -1, -1, -1, -1},
		{0, 4, 5, 1, 0, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{8, 4, 5, 8, 5, 3, 9, 0, 5, 0, 3, 5, -1, -1, -1, -1},
		{9, 4, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{4, 11, 7, 4, 9, 11, 9, 10, 11, -1, -1, -1, -1, -1, -1, -1},
		{0, 8, 3, 4, 9, 7, 9, 11, 7, 9, 10, 11, -1, -1, -1, -1},
		{1, 10, 11, 1, 11, 4, 1, 4, 0, 7, 4, 11, -1, -1, -1, -1},
		{3, 1, 4, 3, 4, 8, 1, 10, 4, 7, 4, 11, 10, 11, 4, -1},
		{4, 11, 7, 9, 11, 4, 9, 2, 11, 9, 1, 2, -1, -1, -1, -1},
		{9, 7, 4, 9, 11, 7, 9, 1, 11, 2, 11, 1, 0, 8, 3, -1},
		{11, 7, 4, 11, 4, 2, 2, 4, 0, -1, -1, -1, -1, -1, -1, -1},
		{11, 7, 4, 11, 4, 2, 8, 3, 4, 3, 2, 4, -1, -1, -1, -1},
		{2, 9, 10, 2, 7, 9, 2, 3, 7, 7, 4, 9, -1, -1, -1, -1},
		{9, 10, 7, 9, 7, 4, 10, 2, 7, 8, 7, 0, 2, 0, 7, -1},
		{3, 7, 10, 3, 10, 2, 7, 4, 10, 1, 10, 0, 4, 0, 10, -1},
		{1, 10, 2, 8, 7, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{4, 9, 1, 4, 1, 7, 7, 1, 3, -1, -1, -1, -1, -1, -1, -1},
		{4, 9, 1, 4, 1, 7, 0, 8, 1, 8, 7, 1, -1, -1, -1, -1},
		{4, 0, 3, 7, 4, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{4, 8, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{9, 10, 8, 10, 11, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{3, 0, 9, 3, 9, 11, 11, 9, 10, -1, -1, -1, -1, -1, -1, -1},
		{0, 1, 10, 0, 10, 8, 8, 10, 11, -1, -1, -1, -1, -1, -1, -1},
		{3, 1, 10, 11, 3, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{1, 2, 11, 1, 11, 9, 9, 11, 8, -1, -1, -1, -1, -1, -1, -1},
		{3, 0, 9, 3, 9, 11, 1, 2, 9, 2, 11, 9, -1, -1, -1, -1},
		{0, 2, 11, 8, 0, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{3, 2, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{2, 3, 8, 2, 8, 10, 10, 8, 9, -1, -1, -1, -1, -1, -1, -1},
		{9, 10, 2, 0, 9, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{2, 3, 8, 2, 8, 10, 0, 1, 8, 1, 10, 8, -1, -1, -1, -1},
		{1, 10, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{1, 3, 8, 9, 1, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{0, 9, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{0, 3, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1}
	};

	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t z = blockIdx.z * blockDim.z + threadIdx.z;
	if (x>=res3i.x()-1 || y>=res3i.y()-1 || z>=res3i.z()-1) return;
	uint32_t res1 = res3i.x();
	uint32_t res2 = res3i.x() * res3i.y();
	uint32_t res3 = res3i.x() * res3i.y() * res3i.z();
	uint32_t idx = x + y * res3i.x() + z * res2;

	uint32_t idx_x=idx;
	uint32_t idx_y=idx + res3;
	uint32_t idx_z=idx + res3 * 2;

	int mask=0;
	if (density[idx]>thresh) mask|=1;
	if (density[idx+1]>thresh) mask|=2;
	if (density[idx+1+res1]>thresh) mask|=4;
	if (density[idx+res1]>thresh) mask|=8;
	idx+=res2;
	if (density[idx]>thresh) mask|=16;
	if (density[idx+1]>thresh) mask|=32;
	if (density[idx+1+res1]>thresh) mask|=64;
	if (density[idx+res1]>thresh) mask|=128;
	idx-=res2;

	if (!mask || mask==255) return;
	int local_edges[12];
	if (vertidx_grid) {
		local_edges[0]=vertidx_grid[idx_x];
		local_edges[1]=vertidx_grid[idx_y+1];
		local_edges[2]=vertidx_grid[idx_x+res1];
		local_edges[3]=vertidx_grid[idx_y];

		local_edges[4]=vertidx_grid[idx_x+res2];
		local_edges[5]=vertidx_grid[idx_y+1+res2];
		local_edges[6]=vertidx_grid[idx_x+res1+res2];
		local_edges[7]=vertidx_grid[idx_y+res2];

		local_edges[8]=vertidx_grid[idx_z];
		local_edges[9]=vertidx_grid[idx_z+1];
		local_edges[10]=vertidx_grid[idx_z+1+res1];
		local_edges[11]=vertidx_grid[idx_z+res1];
	}
	uint32_t tricount=0;
	const int8_t *triangles=triangle_table[mask];
	for (;tricount<15;tricount+=3) if (triangles[tricount]<0) break;
	uint32_t tidx = atomicAdd(counters+1,tricount);
	if (indices_out) {
		for (int i=0;i<15;++i) {
			int j = triangles[i];
			if (j<0) break;
			if (!local_edges[j]) {
				printf("at %d %d %d, mask is %d, j is %d, local_edges is 0\n", x,y,z,mask,j);
			}
			indices_out[tidx+i]=local_edges[j]-1;
		}
	}
}

__global__ void accumulate_1ring(uint32_t num_tris, const uint32_t* indices, const Eigen::Vector3f* verts_in, Eigen::Vector4f* verts_out, Eigen::Vector3f *normals_out) {
	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i>=num_tris) return;
	uint32_t ia=indices[i*3+0];
	uint32_t ib=indices[i*3+1];
	uint32_t ic=indices[i*3+2];
	Eigen::Vector3f pa=verts_in[ia];
	Eigen::Vector3f pb=verts_in[ib];
	Eigen::Vector3f pc=verts_in[ic];

	atomicAdd(&verts_out[ia][0], pb.x()+pc.x());
	atomicAdd(&verts_out[ia][1], pb.y()+pc.y());
	atomicAdd(&verts_out[ia][2], pb.z()+pc.z());
	atomicAdd(&verts_out[ia][3], 2.f);
	atomicAdd(&verts_out[ib][0], pa.x()+pc.x());
	atomicAdd(&verts_out[ib][1], pa.y()+pc.y());
	atomicAdd(&verts_out[ib][2], pa.z()+pc.z());
	atomicAdd(&verts_out[ib][3], 2.f);
	atomicAdd(&verts_out[ic][0], pb.x()+pa.x());
	atomicAdd(&verts_out[ic][1], pb.y()+pa.y());
	atomicAdd(&verts_out[ic][2], pb.z()+pa.z());
	atomicAdd(&verts_out[ic][3], 2.f);

	if (normals_out) {
		Eigen::Vector3f n= (pb-pa).cross(pa-pc); // don't normalise so it's weighted by area
		atomicAdd(&normals_out[ia][0], n.x());
		atomicAdd(&normals_out[ia][1], n.y());
		atomicAdd(&normals_out[ia][2], n.z());
		atomicAdd(&normals_out[ib][0], n.x());
		atomicAdd(&normals_out[ib][1], n.y());
		atomicAdd(&normals_out[ib][2], n.z());
		atomicAdd(&normals_out[ic][0], n.x());
		atomicAdd(&normals_out[ic][1], n.y());
		atomicAdd(&normals_out[ic][2], n.z());
	}
}

void MarchingCubes(const BoundingBox box,const Eigen::Vector3i res3i,const float thresh,const tcnn::GPUMemory<float>& density,const std::string save_density,tcnn::GPUMemory<Eigen::Vector3f>& verts_out,tcnn::GPUMemory<uint32_t>& indices_out,hipStream_t pStream)
{
	tcnn::GPUMemory<uint32_t> counters;

	counters.enlarge(4);
	counters.memset(0);

	size_t n_bytes = res3i.x() * (size_t)res3i.y() * res3i.z() * 3 * sizeof(int);
	auto workspace = tcnn::allocate_workspace(pStream, n_bytes);
	CUDA_CHECK_THROW(hipMemsetAsync(workspace.data(), -1, n_bytes, pStream));

	int* vertex_grid = (int*)workspace.data();

	const dim3 threads = { 4, 4, 4 };
	const dim3 blocks = { tcnn::div_round_up((uint32_t)res3i.x(), threads.x), tcnn::div_round_up((uint32_t)res3i.y(), threads.y), tcnn::div_round_up((uint32_t)res3i.z(), threads.z) };
	// count only
	CUDA_CHECK_THROW(hipStreamSynchronize(hipStreamPerThread));
	gen_vertices<<<blocks, threads, 0,pStream>>>(box, res3i, density.data(), nullptr, nullptr, thresh, counters.data());
	gen_faces<<<blocks, threads,0,pStream>>>(res3i, density.data(), nullptr, nullptr, thresh, counters.data());

	// also just store density data as a ply file
	if (save_density != "") {
		std::vector<float> cpu_density; cpu_density.resize(res3i.x()*res3i.y()*res3i.z());
		density.copy_to_host(cpu_density);

		FILE* plyfile = fopen(save_density.c_str(),"w");
		if (!plyfile) {
			throw std::runtime_error{"Failed to open " + std::string(save_density) + " for writing."};
		}
		fprintf(plyfile,"ply\nformat ascii 1.0\nelement vertex %d\nproperty float x\nproperty float y\nproperty float z\nproperty float density\nend_header\n",res3i.x()*res3i.y()*res3i.z());

		for (uint32_t z=0;z<res3i.z();++z) {
			for (uint32_t y=0;y<res3i.y();++y) {
				for (uint32_t x=0;x<res3i.x();++x) {
					uint32_t idx = x + y * res3i.x() + z * res3i.x() * res3i.y();
					fprintf(plyfile,"%0.5f %0.5f %0.5f %0.5f\n", x/(float)res3i.x(), y/(float)res3i.y(), z/(float)res3i.z(), cpu_density[idx]);
				}
			}
		}
		fclose(plyfile);
	}

	CUDA_CHECK_THROW(hipStreamSynchronize(pStream));
	std::vector<uint32_t> cpucounters; cpucounters.resize(4);
	counters.copy_to_host(cpucounters);
	CUDA_CHECK_THROW(hipStreamSynchronize(hipStreamPerThread));

	//std::cout << "#vertices=" << cpucounters[0] << " #triangles=" << (cpucounters[1]/3)<<std::endl;

	uint32_t n_verts=(cpucounters[0]+127)&~127; // round for later nn stuff
	verts_out.resize(n_verts);
	verts_out.memset(0);
	indices_out.resize(cpucounters[1]);
	CUDA_CHECK_THROW(hipStreamSynchronize(hipStreamPerThread));
	// actually generate verts
	gen_vertices<<<blocks, threads, 0,pStream>>>(box,res3i, density.data(), vertex_grid, verts_out.data(), thresh, counters.data()+2);
	gen_faces<<<blocks, threads,0,pStream>>>(res3i, density.data(), vertex_grid, indices_out.data(), thresh, counters.data()+2);

	CUDA_CHECK_THROW(hipStreamSynchronize(pStream));
}

void save_mesh(
	tcnn::GPUMemory<Eigen::Vector3f>& verts,
	tcnn::GPUMemory<Eigen::Vector3f>& normals,
	tcnn::GPUMemory<Eigen::Vector3f>& colors,
	tcnn::GPUMemory<uint32_t>& indices,
	const char* outputname,
	bool unwrap_it,
	float nerf_scale,
	Eigen::Vector3f nerf_offset
) 
{
	std::vector<Eigen::Vector3f> cpuverts; cpuverts.resize(verts.size());
	std::vector<Eigen::Vector3f> cpunormals; cpunormals.resize(normals.size());
	std::vector<Eigen::Vector3f> cpucolors; cpucolors.resize(colors.size());
	std::vector<uint32_t> cpuindices; cpuindices.resize(indices.size());
	verts.copy_to_host(cpuverts);
	normals.copy_to_host(cpunormals);
	colors.copy_to_host(cpucolors);
	indices.copy_to_host(cpuindices);

	uint32_t numquads = ((cpuindices.size()/3)+1)/2;
	uint32_t numquadsx = uint32_t(sqrtf(numquads)+4) & (~3);
	uint32_t numquadsy = (numquads+numquadsx-1)/numquadsx;
	uint32_t quadresy = 8;
	uint32_t quadresx = quadresy+3;
	uint32_t texw = quadresx*numquadsx;
	uint32_t texh = quadresy*numquadsy;

	if (unwrap_it) {
		uint8_t* tex = (uint8_t*)malloc(texw*texh*3);
		for (uint32_t y = 0; y < texh; ++y) {
			for (uint32_t x = 0; x < texw; ++x) {
				uint32_t q = (x/quadresx)+(y/quadresy)*numquadsx;
				// 0 x x 3 - - 4
				// | .\x x\. . |
				// | . .\x x\. |
				// 2 - - 1 x x 5
				uint32_t xi = x % quadresx, yi = y % quadresy;
				uint32_t t = q*2 + (xi>yi+1);
				int r = (t*923)&255;
				int g = (t*3572)&255;
				int b = (t*5423)&255;
				//if (xi==yi+1 || xi==yi+2)
				//	r=g=b=0;
				tex[x*3+y*3*texw+0]=r;
				tex[x*3+y*3*texw+1]=g;
				tex[x*3+y*3*texw+2]=b;
			}
		}
		std::string Soutputname(outputname);
		Soutputname.substr(Soutputname.size()-4) = std::string(".tga");
		stbi_write_tga(Soutputname.c_str(), texw, texh, 3, tex);
		free(tex);
	}

	FILE* f = fopen(outputname,"wb");
	if (!f) {
		throw std::runtime_error{"Failed to open " + std::string(outputname) + " for writing."};
	}

	std::string Soutputname(outputname);
	if (Soutputname.substr(Soutputname.size()-3) == "ply") {
		// ply file
		fprintf(f,
			"ply\n"
			"format ascii 1.0\n"
			"comment output from https://github.com/NVlabs/instant-ngp\n"
			"element vertex %u\n"
			"property float x\n"
			"property float y\n"
			"property float z\n"
			"property float nx\n"
			"property float ny\n"
			"property float nz\n"
			"property uchar red\n"
			"property uchar green\n"
			"property uchar blue\n"
			"element face %u\n"
			"property list uchar int vertex_index\n"
			"end_header\n"
			, (unsigned int)cpuverts.size()
			, (unsigned int)cpuindices.size()/3
		);
		for (size_t i=0;i<cpuverts.size();++i) 
		{
			Eigen::Vector3f p=(cpuverts[i]-nerf_offset)/nerf_scale;
			Eigen::Vector3f c=cpucolors[i];
			Eigen::Vector3f n=cpunormals[i].normalized();
			unsigned char c8[3]={(unsigned char)tcnn::clamp(c.x()*255.f,0.f,255.f),(unsigned char)tcnn::clamp(c.y()*255.f,0.f,255.f),(unsigned char)tcnn::clamp(c.z()*255.f,0.f,255.f)};
			fprintf(f,"%0.5f %0.5f %0.5f %0.3f %0.3f %0.3f %d %d %d\n", p.x(), p.y(), p.z(), n.x(), n.y(), n.z(), c8[0], c8[1], c8[2]);
		}
		for (size_t i=0;i<cpuindices.size();i+=3) {
			fprintf(f,"3 %d %d %d\n", cpuindices[i+2], cpuindices[i+1], cpuindices[i+0]);
		}
	} else {
		// obj file
		if (unwrap_it) {
			fprintf(f, "mtllib nerf.mtl\n");
		}
		for (size_t i = 0; i < cpuverts.size(); ++i) {
			Eigen::Vector3f p = (cpuverts[i]-nerf_offset)/nerf_scale;
			Eigen::Vector3f c = cpucolors[i];
			fprintf(f,"v %0.5f %0.5f %0.5f %0.3f %0.3f %0.3f\n", p.x(), p.y(), p.z(), tcnn::clamp(c.x(), 0.f, 1.f), tcnn::clamp(c.y(), 0.f, 1.f), tcnn::clamp(c.z(), 0.f, 1.f));
		}
		for (auto &v: cpunormals) {
			auto n = v.normalized();
			fprintf(f,"vn %0.5f %0.5f %0.5f\n", n.x(), n.y(), n.z());
		}
		if (unwrap_it) {
			for (size_t i = 0; i < cpuindices.size(); i++) {
				uint32_t q = (uint32_t)(i/6);
				uint32_t x = (q%numquadsx)*quadresx;
				uint32_t y = (q/numquadsx)*quadresy;
				uint32_t d = quadresy-1;
				switch (i % 6) {
					case 0: break;
					case 1: x += d; y += d; break;
					case 2: y += d; break;
					case 3: x += 3; break;
					case 4: x += 3+d; break;
					case 5: x += 3+d; y += d; break;
				}
				fprintf(f,"vt %0.5f %0.5f\n", ((float)x+0.5f)/float(texw), 1.f-((float)y+0.5f)/float(texh));
			}
			fprintf(f, "g default\nusemtl nerf\ns 1\n");
			for (size_t i = 0; i < cpuindices.size(); i += 3) {
				fprintf(f,"f %u/%u/%u %u/%u/%u %u/%u/%u\n",
					cpuindices[i+2]+1,(uint32_t)i+3,  cpuindices[i+2]+1,
					cpuindices[i+1]+1,(uint32_t)i+2,cpuindices[i+1]+1,
					cpuindices[i+0]+1,(uint32_t)i+1,cpuindices[i+0]+1
				);
			}
		} else {
			for (size_t i = 0; i < cpuindices.size(); i += 3) {
				fprintf(f,"f %u//%u %u//%u %u//%u\n",
					cpuindices[i+2]+1, cpuindices[i+2]+1, cpuindices[i+1]+1, cpuindices[i+1]+1, cpuindices[i+0]+1, cpuindices[i+0]+1
				);
			}
		}
	}
	fclose(f);
}

void compute_mesh_1ring(const tcnn::GPUMemory<Eigen::Vector3f>& verts, const tcnn::GPUMemory<uint32_t>& indices, tcnn::GPUMemory<Eigen::Vector4f>& output_pos, tcnn::GPUMemory<Eigen::Vector3f>& output_normals,hipStream_t pStream)
{
    output_pos.resize(verts.size());
	output_pos.memset(0);
	output_normals.resize(verts.size());
	output_normals.memset(0);
	tcnn::linear_kernel(accumulate_1ring, 0, pStream, indices.size()/3, indices.data(), verts.data(), output_pos.data(), output_normals.data());
	CUDA_CHECK_THROW(hipStreamSynchronize(hipStreamPerThread));
	CUDA_CHECK_THROW(hipStreamSynchronize(pStream));
}


}
